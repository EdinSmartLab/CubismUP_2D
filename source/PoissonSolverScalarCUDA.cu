

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#ifndef _FLOAT_PRECISION_
#define cufftExecFWD hipfftExecR2C
#define cufftExecBWD hipfftExecC2R
#define cufftPlanFWD HIPFFT_R2C
#define cufftPlanBWD HIPFFT_C2R
#define cufftValT hipfftReal
#define cufftCmpT hipfftComplex
#else //_FLOAT_PRECISION_
#define cufftExecFWD cufftExecD2Z
#define cufftExecBWD cufftExecZ2D
#define cufftPlanFWD CUFFT_D2Z
#define cufftPlanBWD CUFFT_Z2D
#define cufftValT cufftDoubleReal
#define cufftCmpT cufftDoubleComplex
#endif//_FLOAT_PRECISION_

void freePlan(hipfftHandle& plan) {
  hipfftDestroy(plan);
}
void makePlan(hipfftHandle& handle, const int mx, const int my, hipfftType plan) {
  hipfftPlan2d(&handle, mx, my, plan);
}
void freeCuMem(cufftValT* buf) {
  hipFree(buf);
}
void allocCuMem(cufftValT* & ptr, const size_t size) {
  hipMalloc((void **)& ptr, size);
}

__global__ void solve(const int mx, const int my, const int my_hat,
  const cufftValT facX, const cufftValT facY, const cufftValT norm,
  cufftCmpT*const in_out)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ( (i >= mx) || (j >= my_hat) ) return;
  const int linidx = i * my_hat + j;
  const int kx = (i <= mx/2) ? i : -(mx-i);
  const int ky = (j <= my/2) ? j : -(my-j);
  const cufftValT rkx = kx*facX, rky = ky*facY;
  const cufftValT kinv = (kx==0 && ky==0) ? 0 : -(cufftValT)1/(rkx*rkx+rky*rky);
  in_out[linidx].x *= norm * kinv;
  in_out[linidx].y *= norm * kinv;
}

void cuSolve(const hipfftHandle&fwd, const hipfftHandle&bwd, const int mx,
  const int my, const cufftValT h, cufftValT*const rhs, cufftValT*const rhs_gpu)
{
  const int my_hat = my/2 +1;
  const cufftValT facX = 2.0*M_PI/(mx*h);
  const cufftValT facY = 2.0*M_PI/(my*h);
  const cufftValT norm = 1./(mx*my);

  hipMemcpy(rhs,rhs_gpu, 2*mx*my_hat*sizeof(cufftValT),hipMemcpyHostToDevice);
  cufftExecFWD(fwd, rhs_gpu, (cufftCmpT*) rhs_gpu);

  dim3 dimBlock(16, 16);
  dim3 dimGrid(mx / dimBlock.x, (my/2) / dimBlock.y + 1);
  solve<<<dimGrid,dimBlock>>> (mx,my,my_hat,facX,facY,norm,(cufftCmpT*)rhs_gpu);

  cufftExecBWD(bwd, (cufftCmpT*) rhs_gpu, rhs_gpu);
  hipMemcpy(rhs_gpu,rhs, 2*mx*my_hat*sizeof(cufftValT),hipMemcpyDeviceToHost);
}


//__global__ void complex2RealScaled(float2 * __restrict__ d_r,
//  float * __restrict__ d_result, const int M, const int N, float scale)
//{
//    const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
//    const int tidy = threadIdx.y + blockIdx.y * blockDim.y;
//    if ((tidx >= M) || (tidy >= N)) return;
//    d_result[tidy * M + tidx] = scale * (d_r[tidy * M + tidx].x - d_r[0].x);
//}
