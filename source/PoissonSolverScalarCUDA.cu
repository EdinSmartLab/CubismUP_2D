#include "hip/hip_runtime.h"
#include "PoissonSolverCuda.h"

__device__ static inline void solve(const int mx,const int my, const int my_hat,
  const cufftValT facX, const cufftValT facY, const cufftValT norm,
  ccufftCmpT*const in_out)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ( (i >= mx) || (j >= my_hat) ) return;
  const int linidx = i * my_hat + j;
  const int kx = (i <= mx/2) ? i : -(mx-i);
  const int ky = (j <= my/2) ? j : -(my-j);
  const Real rkx = kx*facX, rky = ky*facY;
  const Real kinv = (kx==0 && ky==0) ? 0 : -(Real)1/(rkx*rkx+rky*rky);
  in_out[linidx].x *= norm * kinv;
  in_out[linidx].y *= norm * kinv;
}

void PoissonSolverCuda::solve() const
{
  hipMemcpy(rhs, rhs_gpu, 2*mx*my_hat*sizeof(Real), hipMemcpyHostToDevice);
  cufftExecFWD(fwd, rhs_gpu, (cufftCmpT*) rhs_gpu);

  dim3 dimBlock(16, 16);
  dim3 dimGrid(N / dimBlock.x, (my/2) / dimBlock.y + 1);
  solve <<<dimGrid, dimBlock>>> (mx, my, my_hat, facX, facY, norm, rhs_gpu);

  cufftExecBWD(bwd, (cufftCmpT*) rhs_gpu, rhs_gpu);
  hipMemcpy(rhs_gpu, rhs, 2*mx*my_hat*sizeof(Real), hipMemcpyDeviceToHost);

  _fftw2cub();
}

PoissonSolverCuda::PoissonSolverCuda(FluidGrid& _grid, const bool bFrespace):
grid(_grid), mx(bFrespace? 2 * nx - 1 : nx), my(bFrespace? 2 * ny - 1 : ny)
{
  hipfftPlan2d(&fwd, mx, my, HIPFFT_R2C);
  hipfftPlan2d(&bwd, mx, my, HIPFFT_C2R);
  assert(2*sizeof(Real) == sizeof(ccufftCmpT));
  rhs = (Real*) malloc(mx * my_hat * 2 * sizeof(Real) );
  hipMalloc((void **)& gpu_rhs, mx * my_hat * sizeof(ccufftCmpT) );
}


//__global__ void complex2RealScaled(float2 * __restrict__ d_r,
//  float * __restrict__ d_result, const int M, const int N, float scale)
//{
//    const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
//    const int tidy = threadIdx.y + blockIdx.y * blockDim.y;
//    if ((tidx >= M) || (tidy >= N)) return;
//    d_result[tidy * M + tidx] = scale * (d_r[tidy * M + tidx].x - d_r[0].x);
//}
