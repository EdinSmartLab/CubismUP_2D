
#include <hipfft/hipfft.h>
#include <cassert>
#ifndef _FLOAT_PRECISION_
#define cufftExecFWD hipfftExecD2Z
#define cufftExecBWD hipfftExecZ2D
#define cufftPlanFWD HIPFFT_D2Z
#define cufftPlanBWD HIPFFT_Z2D
#define Real hipfftDoubleReal
#define Cmpl hipfftDoubleComplex
#else //_FLOAT_PRECISION_
#define cufftExecFWD cufftExecR2C
#define cufftExecBWD cufftExecC2R
#define cufftPlanFWD CUFFT_R2C
#define cufftPlanBWD CUFFT_C2R
#define Real cufftReal
#define Cmpl cufftComplex
#endif//_FLOAT_PRECISION_

void freePlan(hipfftHandle& plan) {
  hipfftDestroy(plan);
}
void makePlan(hipfftHandle& handle, const int mx, const int my, hipfftType plan) {
  hipfftPlan2d(&handle, mx, my, plan);
}
void freeCuMem(Real* buf) {
  hipFree(buf);
}
void allocCuMem(Real* & ptr, const size_t size) {
  hipMalloc((void **)& ptr, size);
}

__global__ void kPeriodic(const int mx, const int my, const int my_hat,
  const Real facX, const Real facY, const Real norm,
  Cmpl*const in_out) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ( (i >= mx) || (j >= my_hat) ) return;
  const int linidx = i * my_hat + j;
  const int kx = (i <= mx/2) ? i : -(mx-i);
  const int ky = (j <= my/2) ? j : -(my-j);
  const Real rkx = kx*facX, rky = ky*facY;
  const Real kinv = (kx==0 && ky==0) ? 0 : -(Real)1/(rkx*rkx+rky*rky);
  in_out[linidx].x *= norm * kinv;
  in_out[linidx].y *= norm * kinv;
}

__global__ void kFreespace(const int mx, const int my_hat,
  const Real*const G_hat, Cmpl*const in_out) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ( (i >= mx) || (j >= my_hat) ) return;
  const int linidx = i * my_hat + j;
  in_out[linidx].x *= G_hat[linidx];
  in_out[linidx].y *= G_hat[linidx];
}

void dPeriodic(const hipfftHandle&fwd, const hipfftHandle&bwd, const int mx,
 const int my, const Real h, Real*const rhs,Real*const rhs_gpu)
{
  const int my_hat = my/2 +1;
  const Real facX = 2.0*M_PI/(mx*h);
  const Real facY = 2.0*M_PI/(my*h);
  const Real norm = 1./(mx*my);

  hipMemcpy(rhs_gpu,rhs, 2*mx*my_hat*sizeof(Real), hipMemcpyHostToDevice);
  cufftExecFWD(fwd, rhs_gpu, (Cmpl*) rhs_gpu);

  dim3 dimB(16, 16);
  assert(mx % dimB.x == 0);
  assert(my % dimB.y == 0);
  dim3 dimG(mx / dimB.x, my_hat / dimB.y + 1);
  kPeriodic <<<dimG,dimB>>> (mx,my,my_hat, facX,facY,norm, (Cmpl*)rhs_gpu);

  cufftExecBWD(bwd, (Cmpl*) rhs_gpu, rhs_gpu);
  hipMemcpy(rhs,rhs_gpu, 2*mx*my_hat*sizeof(Real), hipMemcpyDeviceToHost);
}

#include "cstdio"
#include <hip/hip_runtime.h>

class GpuTimer {
    hipEvent_t B, E; //cudaEventBlockingSync
  public:
    GpuTimer() { hipEventCreate(&B); hipEventCreate(&E); }
    GpuTimer(int flag) { hipEventCreateWithFlags(&B, flag); hipEventCreateWithFlags(&E, flag); }
    ~GpuTimer() { hipEventDestroy(B); hipEventDestroy(E); }
    void start() { hipEventRecord(B, 0); }
    void stop() { hipEventRecord(E, 0); }
    float get() {
      float elapsed;
      hipEventSynchronize(E);
      hipEventElapsedTime(&elapsed, B, E);
      return elapsed;
    }
};

void dFreespace(const hipfftHandle&fwd, const hipfftHandle&bwd, const int nx,
  const int ny, Real*const rhs, const Real*const G_hat, Real*const rhs_gpu)
{
  const int mx = 2 * nx - 1, my = 2 * ny - 1;
  const int my_hat = my/2 +1, ny_hat = ny/2 +1;
  //GpuTimer t0, t1, t2, t3, t4;
  //t0.start();
  hipMemcpy2D(rhs_gpu,2*my_hat*sizeof(Real), rhs,2*ny_hat*sizeof(Real),
    ny*sizeof(Real), nx, hipMemcpyHostToDevice);
  //t0.stop();
  //t1.start();
  cufftExecFWD(fwd, rhs_gpu, (Cmpl*) rhs_gpu);
  //t1.stop();
  //t2.start();
  dim3 dimB(16, 16);
  assert((mx+1) % dimB.x == 0);
  assert(my_hat % dimB.y == 0);
  dim3 dimG( (mx+1) / dimB.x, my_hat / dimB.y);
  kFreespace <<<dimG,dimB>>> (mx,my_hat, G_hat, (Cmpl*)rhs_gpu);
  //t2.stop();
  //t3.start();
  cufftExecBWD(bwd, (Cmpl*) rhs_gpu, rhs_gpu);
  //t3.stop();
  //t4.start();
  hipMemcpy2D(rhs,2*ny_hat*sizeof(Real), rhs_gpu,2*my_hat*sizeof(Real),
    ny*sizeof(Real), nx, hipMemcpyDeviceToHost);
  hipMemset(rhs_gpu, 0, mx*my_hat * 2*sizeof(Real) );
  //t4.stop();
  //printf("%f %f %f %f %f\n",t0.get(),t1.get(),t2.get(),t3.get(),t4.get());
}

__global__ void kGreen(const int nx, const int ny, const int mx, const int my,
  const int my_hat, const Real fac, const Real h, Real*const in_out) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ( (i >= mx) || (j >= my) ) return;
  const int linidx = j + 2*my_hat*i;
  const Real xi = i>=nx? mx-i : i;
  const Real yi = j>=ny? my-j : j;
  const Real r = std::sqrt(xi*xi + yi*yi);
  if(r > 0) in_out[linidx] = fac * std::log(h * r);
  // r_eq = h / sqrt(pi)
  // G = 1/4 * r_eq^2 * (2* ln(r_eq) - 1)
  else      in_out[linidx] = fac/2 * (2*std::log(h/std::sqrt(M_PI)) - 1);
}

__global__ void kCopyC2R(const int mx, const int my_hat, const Real norm,
  const Cmpl*const G_hat, Real*const m_kernel) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ( (i >= mx) || (j >= my_hat) ) return;
  const int linidx = j + my_hat*i;
  m_kernel[linidx] = G_hat[linidx].x * norm;
}

void clearCuMem(Real * buf, const size_t size) { hipMemset(buf, 0, size ); }

void initGreen(const int nx, const int ny, const Real h, Real*const m_kernel)
{
  const int mx = 2 * nx - 1, my = 2 * ny - 1;
  const int my_hat = my/2 +1;
  Real * tmp;
  hipMalloc((void **)& tmp, mx * my_hat * sizeof(Cmpl) );
  {
    const Real fac = h * h / ( 2*M_PI );
    dim3 dimB(16, 16);
    assert((mx+1) % dimB.x == 0);
    assert((my+1) % dimB.y == 0);
    dim3 dimG( (mx+1) / dimB.x, (my+1) / dimB.y);
    kGreen<<<dimG, dimB>>> (nx,ny, mx,my, my_hat, fac, h, tmp);
  }
  {
    hipfftHandle fwd;
    hipfftPlan2d(&fwd, mx, my, cufftPlanFWD);
    cufftExecFWD(fwd, tmp, (Cmpl*) tmp);
    hipfftDestroy(fwd);
  }
  {
    const Real norm = 1.0 / (mx * my);
    dim3 dimB(16, 16);
    assert((mx+1) % dimB.x == 0);
    assert(my_hat % dimB.y == 0);
    dim3 dimG( (mx+1) / dimB.x, my_hat / dimB.y);
    kCopyC2R<<<dimG, dimB>>> (mx, my_hat, norm, (Cmpl*)tmp, m_kernel);
  }
  hipFree(tmp);
}
