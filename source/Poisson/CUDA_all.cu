//
//  CubismUP_2D
//  Copyright (c) 2018 CSE-Lab, ETH Zurich, Switzerland.
//  Distributed under the terms of the MIT license.
//
//  Created by Guido Novati (novatig@ethz.ch).
//


#include <hipfft/hipfft.h>
#include <cassert>
#ifndef _FLOAT_PRECISION_
#define cufftExecFWD hipfftExecD2Z
#define cufftExecBWD hipfftExecZ2D
#define cufftPlanFWD HIPFFT_D2Z
#define cufftPlanBWD HIPFFT_Z2D
#define Real hipfftDoubleReal
#define Cmpl hipfftDoubleComplex
#else //_FLOAT_PRECISION_
#define cufftExecFWD cufftExecR2C
#define cufftExecBWD cufftExecC2R
#define cufftPlanFWD CUFFT_R2C
#define cufftPlanBWD CUFFT_C2R
#define Real cufftReal
#define Cmpl cufftComplex
#endif//_FLOAT_PRECISION_

void freePlan(hipfftHandle& plan) {
  hipfftDestroy(plan);
}
void makePlan(hipfftHandle& handle, const int MY, const int MX, hipfftType plan) {
  hipfftPlan2d(&handle, MY, MX, plan);
}
void freeCuMem(Real* buf) {
  hipFree(buf);
}
void allocCuMem(Real* & ptr, const size_t size) {
  hipMalloc((void **)& ptr, size);
}

__global__ void kPeriodic(const int MY, const int MX, const int MX_hat,
  const Real facX, const Real facY, const Real norm, Cmpl*const in_out)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ( (j >= MY) || (i >= MX_hat) ) return;
  const int kx = (i<=MX/2) ? i : -(MX-i);
  const int ky = (j<=MY/2) ? j : -(MY-j);
  const Real rkx = kx*facX, rky = ky*facY;
  const Real kinv = (kx==0 && ky==0) ? 0 : -(Real)1/(rkx*rkx + rky*rky);
  in_out[j * MX_hat + i].x *= norm * kinv;
  in_out[j * MX_hat + i].y *= norm * kinv;
}

__global__ void kFreespace(const int MY, const int MX_hat,
  const Real*const G_hat, Cmpl*const in_out)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ( (j >= MY) || (i >= MX_hat) ) return;
  in_out[j * MX_hat + i].x *= G_hat[j * MX_hat + i];
  in_out[j * MX_hat + i].y *= G_hat[j * MX_hat + i];
}

void dPeriodic(const hipfftHandle&fwd, const hipfftHandle&bwd, const int MY,
 const int MX, const Real h, Real*const rhs, Real*const rhs_gpu)
{
  const int MX_hat = MX/2 +1;
  const Real facX = 2*M_PI/MX, facY = 2*M_PI/MY, norm = 1.0/(MX*MY);

  hipMemcpy(rhs_gpu, rhs, 2*MY*MX_hat*sizeof(Real), hipMemcpyHostToDevice);
  cufftExecFWD(fwd, rhs_gpu, (Cmpl*) rhs_gpu);

  dim3 dimB(16, 16), dimG(MX_hat / dimB.x + 1, MY / dimB.y);
  assert((MX % dimB.x == 0) && (MY % dimB.y == 0));
  kPeriodic <<<dimG,dimB>>> (MY, MX, MX_hat, facX,facY,norm, (Cmpl*)rhs_gpu);

  cufftExecBWD(bwd, (Cmpl*) rhs_gpu, rhs_gpu);
  hipMemcpy(rhs,rhs_gpu, 2*MY*MX_hat*sizeof(Real), hipMemcpyDeviceToHost);
}

#include "cstdio"
#include <hip/hip_runtime.h>

class GpuTimer
{
    hipEvent_t B, E; //cudaEventBlockingSync
  public:
    GpuTimer() { hipEventCreate(&B); hipEventCreate(&E); }
    GpuTimer(int flag) { hipEventCreateWithFlags(&B, flag); hipEventCreateWithFlags(&E, flag); }
    ~GpuTimer() { hipEventDestroy(B); hipEventDestroy(E); }
    void start() { hipEventRecord(B, 0); }
    void stop() { hipEventRecord(E, 0); }
    float get() {
      float elapsed;
      hipEventSynchronize(E);
      hipEventElapsedTime(&elapsed, B, E);
      return elapsed;
    }
};

void dFreespace(const hipfftHandle&fwd, const hipfftHandle&bwd, const int NX,
  const int NY, Real*const rhs, const Real*const G_hat, Real*const rhs_gpu)
{
  const int MX = 2 * NX - 1, MY = 2 * NY - 1, MX_hat = MX/2 +1;
  //GpuTimer t0, t1, t2, t3, t4;
  //t0.start();
  hipMemcpy2D(rhs_gpu, 2*MX_hat*sizeof(Real), rhs, NX*sizeof(Real),
    NX*sizeof(Real), NY, hipMemcpyHostToDevice);
  //t0.stop();
  //t1.start();
  cufftExecFWD(fwd, rhs_gpu, (Cmpl*) rhs_gpu);
  //t1.stop();
  //t2.start();
  dim3 dimB(16, 16), dimG( MX_hat / dimB.x, (MY+1) / dimB.y);
  assert( ((MY+1) % dimB.y) == 0 && (MX_hat % dimB.x) == 0 );
  kFreespace <<<dimG,dimB>>> (MY, MX_hat, G_hat, (Cmpl*)rhs_gpu);
  //t2.stop();
  //t3.start();
  cufftExecBWD(bwd, (Cmpl*) rhs_gpu, rhs_gpu);
  //t3.stop();
  //t4.start();
  hipMemcpy2D(rhs, NX*sizeof(Real), rhs_gpu, 2*MX_hat*sizeof(Real),
    NX*sizeof(Real), NY, hipMemcpyDeviceToHost);
  hipMemset(rhs_gpu, 0, MY * MX_hat * 2 * sizeof(Real) );
  //t4.stop();
  //printf("%f %f %f %f %f\n",t0.get(),t1.get(),t2.get(),t3.get(),t4.get());
}

__global__ void kGreen(const int NX, const int NY, const int MX, const int MY,
  const int MX_hat, const Real fac, const Real h, Real*const in_out)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ( (i >= MX) || (j >= MY) ) return;
  const Real xi = i>=NX? MX-i : i, yi = j>=NY? MY-j : j;
  const Real r = std::sqrt(xi*xi + yi*yi);
  if(r > 0) in_out[i +2*MX_hat*j] = fac * std::log(h * r);
  // r_eq = h / sqrt(pi)
  // G = 1/4 * r_eq^2 * (2* ln(r_eq) - 1)
  else      in_out[i +2*MX_hat*j] = fac/2 * (2*std::log(h/std::sqrt(M_PI)) - 1);
}

__global__ void kCopyC2R(const int MY, const int MX_hat, const Real norm,
  const Cmpl*const G_hat, Real*const m_kernel)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ( (i >= MX_hat) || (j >= MY) ) return;
  m_kernel[i + MX_hat*j] = G_hat[i + MX_hat*j].x * norm;
}

void clearCuMem(Real * buf, const size_t size) { hipMemset(buf, 0, size ); }

void initGreen(const int NY, const int NX, const Real h, Real*const m_kernel)
{
  const int MX = 2 * NX - 1, MY = 2 * NY - 1, MX_hat = MX/2 +1;
  Real * tmp;
  hipMalloc((void **)& tmp, MY * MX_hat * sizeof(Cmpl) );
  {
    const Real fac = 1 / ( 2*M_PI );
    dim3 dimB(16, 16), dimG( (MX+1) / dimB.x, (MY+1) / dimB.y);
    assert(((MX+1) % dimB.x == 0) && ((MY+1) % dimB.y == 0));
    kGreen<<<dimG, dimB>>> (NX,NY, MX,MY, MX_hat, fac, h, tmp);
  }
  {
    hipfftHandle fwd;
    hipfftPlan2d(&fwd, MY, MX, cufftPlanFWD);
    cufftExecFWD(fwd, tmp, (Cmpl*) tmp);
    hipfftDestroy(fwd);
  }
  {
    const Real norm = 1.0 / (MX * MY);
    dim3 dimB(16, 16), dimG( MX_hat / dimB.x, (MY+1) / dimB.y);
    assert((MX_hat % dimB.x == 0) && ((MY+1) % dimB.y == 0));
    kCopyC2R<<<dimG, dimB>>> (MY, MX_hat, norm, (Cmpl*)tmp, m_kernel);
  }
  hipFree(tmp);
}
